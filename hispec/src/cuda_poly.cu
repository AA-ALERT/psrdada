#include "hip/hip_runtime.h"
#if USE_DADA
#include <ipcio.h>
#include <dada_hdu.h>
#endif
#include <sys/time.h>
#include <hipfft/hipfft.h>
#include <complex.h>

#include "cuda_poly.h"

/********************************************************************
 *                     Various Kernel Functions                     *
 * *****************************************************************/

/* Kernel function for polyphase filter, using the overlap-add method */
__global__ void overlap_add_kernel(float *out, float *in, float *window, int windowBlocks)
{
  int i;

  int size = blockDim.x * gridDim.x;

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  int batch = blockIdx.y;
  int nbatch = gridDim.y;

  int inp = blockIdx.z;


  for( i = 0; i < windowBlocks; i++ )
  {
    out[inp*nbatch*size + batch*size + index] = 
      out[inp*nbatch*size + batch*size + index] + window[i*size + index] * 
      in[inp*(nbatch+windowBlocks-1)*size + (batch+i)*size + index];
  }
}

#if 0 /* Newer but somehow slower method */
__global__ void overlap_add_kernel(float *out, float *in, float *window)
{
  int size = blockDim.x * gridDim.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  int batch = blockIdx.y;
  int nbatch = gridDim.y;

  int inp = blockIdx.z;
  int windowBlocks = blockDim.y;

  /* Use shared memory to store the intermiediate results */
  /* FIXME: Currently for simplicity, hard code the share memory size.
   * It should be the same as numThreads * windowBlocks.
   */
  int tempIdx = threadIdx.x*blockDim.y + threadIdx.y;
  __shared__ float temp[1024];

  temp[tempIdx] = 
    window[threadIdx.y*size + index] * 
    in[inp*(nbatch+windowBlocks-1)*size + (batch+threadIdx.y)*size + index];

  __syncthreads();
  /* FIXME: Currently hard coded for 8 windowBlocks. 
   * Should make it more rebust with a loop, or use another level 
   * of parallel sum reduction 
   */
  if( threadIdx.y % 2 == 0 )
    temp[tempIdx] += temp[tempIdx+1];

  __syncthreads();

  if( threadIdx.y % windowBlocks == 0 )
    out[inp*nbatch*size + batch*size + index] = 
      temp[tempIdx] + temp[tempIdx+2] + temp[tempIdx+4] + temp[tempIdx+6];
}
#endif
/* Kernel function for polyphase filter, using oversample-decimate method */
/* FIXME */
__global__ void oversample_decimate_kernel(float *out, float *in, float *window, int windowBlocks)
{

}


/* Kernel for reading unsigned data into GPU */
__global__ void unpackUnsignedData_kernel(unsigned char *buf, float *out)
{
  int npoints = blockDim.x * gridDim.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  int nbatch = gridDim.y;
  int batch = blockIdx.y; 
  
  int ninp = gridDim.z;
  int inp = blockIdx.z; 

  out[inp*nbatch*npoints + batch*npoints + index] = 
    (float)( buf[batch*npoints*ninp + index*ninp + inp] - 128 );
}

/* Kernel for reading signed data into GPU */
/* FIXME */
__global__ void unpackSignedData_kernel(unsigned char *buf, float *out)
{
  int npoints = blockDim.x * gridDim.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  int nbatch = gridDim.y;
  int batch = blockIdx.y;

  int ninp = gridDim.z;
  int inp = blockIdx.z;

  out[inp*nbatch*npoints + batch*npoints + index] = 
    ( (char *)buf )[batch*npoints*ninp + index*ninp + inp];
}

/* Kernel for performing CMAC auto correlation */
__global__ void CMAC_auto_kernel(int nbatch, hipfftComplex *ft, float *corr)
{
  int batch;
  int nchan = blockDim.x * gridDim.x;
  int inp = blockIdx.y;
  int chan = blockIdx.x * blockDim.x + threadIdx.x;
  int index;

  for( batch = 0; batch < nbatch; batch++ )
  {
    /* The fourier transform is 1 size larger */
    index = inp*nbatch*(nchan+1) + batch*(nchan+1) + chan;

    corr[inp*nchan + chan] +=
      ft[index].x * ft[index].x + ft[index].y * ft[index].y; 
  }
}

/* Kernel for performing CMAC cross correlation */
__global__ void CMAC_cross_kernel(int ninp, int nbatch, hipfftComplex *ft, hipfftComplex *corr)
{
  int i, j;
  int batch;
  
  int nchan = blockDim.x * gridDim.x;
  int cross = blockIdx.y;
  
  int chan = blockIdx.x * blockDim.x + threadIdx.x;
  int inp1 = -1, inp2 = 1;

  int index1, index2;

  /* An algorithm to calculate inp1 and inp2 in this thread for cross correlation. */
  /* Need a bit more experiments to investigate the time performance of this method */  
  i = -1;
  j = -1;
  do
  {
    i++;
    inp1++;
    inp2 = inp1 + 1;
    j = j + ninp - inp1 - 1;
    while( i < cross && i < j )
    {
      inp2++;
      i++;
    }
  }
  while( i < cross );

  for( batch = 0; batch < nbatch; batch++ )
  {
    /* Calculate the indices from inp1 and inp2 to identify which input to be used */
    index1 = inp1*nbatch*nchan + batch*nchan + chan;
    index2 = inp2*nbatch*nchan + batch*nchan + chan;

    corr[cross * nchan + chan].x += 
      ft[index1].x * ft[index2].x + ft[index1].y * ft[index2].y;
    
    corr[cross * nchan + chan].y += 
      ft[index1].y * ft[index2].x - ft[index1].x * ft[index2].y;
  }
}

/* Kernel for performing the creal function for auto correlation 
 * (extracting real part of complex array from the needed stream) */
#if 0 /* Not needed anymore as the auto correlation now uses float * directly */
__global__ void creal_auto_kernel(float *out, hipfftComplex *buf)
{
  int i;
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int ntx = blockDim.x;
  int nchan = ntx * gridDim.x;
  int cprod = 0;

  int chan = bx * ntx + tx;

  int ninp = gridDim.y;

  int index = by * nchan + chan;

  /* An algorithm to calculate cprod for this thread, only for auto correlation */
  for( i = 0; i < by; i++ )
    cprod += ninp - i;

  out[index] = buf[cprod * nchan + chan].x;
}
#endif

/* Kernel for performing normalisation */
__global__ void normalise_complex_kernel(hipfftComplex *buf, float normaliser)
{
  int nchan = blockDim.x * gridDim.x;
  int index = blockIdx.y * nchan + blockIdx.x * blockDim.x + threadIdx.x;
  
  buf[index].x *= normaliser;
  buf[index].y *= normaliser;
}

__global__ void normalise_float_kernel(float *buf, float normaliser)
{
  int nchan = blockDim.x * gridDim.x;
  int index = blockIdx.y * nchan + blockIdx.x * blockDim.x + threadIdx.x;

  buf[index] *= normaliser;
}


/********************************************************************
 * **********              End of GPU kernels               *********
 *******************************************************************/



/* GPU correlator, will replace the CPU correlator if GPU is enabled */
void gpu_corr( int nchan, int ninp, int ncross, int windowBlocks, int nbatch,
    int prod_type, char *polyMethod, float *cuda_inp_buf, float *cuda_window_buf, 
    float *cuda_poly_buf, hipfftComplex *cuda_ft_buf, 
    hipfftComplex *cuda_cross_corr, float *cuda_auto_corr,
    float *poly_time, float *fft_time, float *cmac_time )
{
  struct timeval thetime;

  /* Multiply the data with the window function */
  gettimeofday(&thetime, NULL);
  polyphase_gpu( ninp, windowBlocks, nchan * 2, nbatch, polyMethod, 
      cuda_poly_buf, cuda_inp_buf, cuda_window_buf);
  hipDeviceSynchronize();
  *poly_time += elapsed_time(&thetime);

  /* Perform CUDA FFT */
  gettimeofday(&thetime, NULL);
  do_CUFFT(nchan, ninp, nbatch, cuda_poly_buf, cuda_ft_buf);
  hipDeviceSynchronize();
  *fft_time += elapsed_time(&thetime);

  /* Perform CMAC */
  gettimeofday(&thetime, NULL);
  do_CUDA_CMAC(nchan, ninp, ncross, nbatch, prod_type, 
      cuda_ft_buf, cuda_cross_corr, cuda_auto_corr);
  hipDeviceSynchronize();
  *cmac_time += elapsed_time(&thetime);
}

/* Reads memory into GPU in batch */
#if USE_DADA
int readDataToGPU(int nchan, int ninp, int windowBlocks, int nbatch, int bits_per_samp, dada_hdu_t *hdu, float *cuda_inp_buf, int debug, int wordtype)
#else
int readDataToGPU(int nchan, int ninp, int windowBlocks, int nbatch, int bits_per_samp, FILE *fpin, float *cuda_inp_buf, int debug, int wordtype)
#endif
{
  int i;
  static int init = 0, ntoread = 0;
  static unsigned char *buffer = NULL;
  static unsigned char *cudaBuffer;

  int nread;
  struct timeval starttime;

  gettimeofday( &starttime, NULL );

  if( init == 0 )
  {
    ntoread = ninp * nchan * 2 * nbatch * bits_per_samp / 8;
    init = 1;
    buffer = (unsigned char *)malloc(ntoread);
    hipMalloc( (void **)&cudaBuffer, ntoread );
    if( debug )
      fprintf( stderr, "size of read buffer: %d bytes\n", ntoread );
  }

  struct timeval thetime;
  float fileReadTime=0, cudaCopyTime=0, totalTime=0, unpackTime=0;

  gettimeofday( &thetime, NULL );
#if USE_DADA
  nread = ipcio_read( hdu->data_block, (char *)buffer, ntoread );
#else
  nread = fread( buffer, 1, ntoread, fpin );
#endif
  fileReadTime += elapsed_time(&thetime);
  

  if( nread < ntoread ) 
  {
    free( buffer );
    hipFree( cudaBuffer );
    return 1; 
  }

  /* Call the appropriate function base of wordtype */
  /* Experiments show that CUDA kernel is slower than CPU with ninp == 1.
   * Still need to test with ninp > 1 */
  
  /*if( ninp == 1 && nbatch == 1 )
  {
    for( chan = 0; chan < nchan * 2; chan++ )
    {
      temp[chan] = (float)(buffer[chan] - 128);
    }
    hipMemcpy(&cuda_inp_buf[nchan*2*tail], temp, nchan * 2 * sizeof(float), 
	hipMemcpyHostToDevice);
  }

  else*/

  gettimeofday( &thetime, NULL );
  hipMemcpy( cudaBuffer, buffer, ntoread, hipMemcpyHostToDevice );
  /* Copy the last (windowBlocks-1) chunks to the beginning for each stream.
   * Do it after the first reading.
   */
  if( init == 1 )
  {
    for( i = 0; i < ninp; i++ )
      hipMemcpy( &cuda_inp_buf[i * (nbatch+windowBlocks-1) * nchan * 2], 
	  &cuda_inp_buf[i*(nbatch+windowBlocks-1)*nchan*2 + (nbatch)*nchan*2], 
	  (windowBlocks-1) * nchan * 2 * sizeof(float), hipMemcpyDeviceToDevice );
  }
  cudaCopyTime += elapsed_time(&thetime);
  
  /* Thread number should be multiple of 32 for best efficiency */
  /* Assume nchan to be power of 2 */
  dim3 threads( 128, 1, 1 );
  dim3 blocks( nchan * 2 / 128, nbatch, ninp );
  
  /* cuda_inp_buf needs to be offset by (windowBlocks-1) chunks due to the circular queue design */
  gettimeofday( &thetime, NULL );
  if( wordtype == 0 )
    unpackUnsignedData_kernel<<< blocks, threads >>>(cudaBuffer, &cuda_inp_buf[(windowBlocks-1) * nchan * 2]);
  /* FIXME: Not sure about the correctness of signed data unpacking */
  else if( wordtype == 1 )
    unpackSignedData_kernel<<< blocks, threads >>>(cudaBuffer, &cuda_inp_buf[(windowBlocks-1) * nchan * 2]);

  hipDeviceSynchronize();
  unpackTime += elapsed_time(&thetime);
  totalTime += elapsed_time(&starttime);

  //fprintf( stderr, "File read: %f, hipMemcpy: %f, data unpack: %f, total: %f\n", 
    //  fileReadTime, cudaCopyTime, unpackTime, totalTime );

  return 0;
}

/* Assume that the digitised data coming in 8 bits array */
/* The data size is assumed to be nchan*2*ninp*nbatch */
int unpackDigitisedDataToGPU(int nchan, int ninp, int windowBlocks, int nbatch, int bits_per_samp, unsigned char *digitised_data, float *cuda_inp_buf, int debug, int wordtype)
{
  int i;
  static int init = 0, ntoread = 0;
  static unsigned char *cudaBuffer;
  //static unsigned char *buffer;
  static int numThreads = 64;

  struct timeval starttime;

  gettimeofday( &starttime, NULL );

  if( init == 0 )
  {
    ntoread = ninp * nchan * 2 * nbatch * bits_per_samp / 8;
    init = 1;
    hipMalloc( (void **)&cudaBuffer, ntoread );
    //buffer = (unsigned char *)malloc(ntoread);
    if( debug )
      fprintf( stderr, "size of read buffer: %d bytes\n", ntoread );
  }

  struct timeval thetime;
  float cudaCopyTime=0, totalTime=0, unpackTime=0;

  //memcpy( buffer, digitised_data, ntoread );

  gettimeofday( &thetime, NULL );
  hipMemcpy( cudaBuffer, digitised_data, ntoread, hipMemcpyHostToDevice );
  cudaCopyTime += elapsed_time(&thetime);
  /* Copy the last (windowBlocks-1) chunks to the beginning for each stream.
   * 
   */
  for( i = 0; i < ninp; i++ )
    hipMemcpy( &cuda_inp_buf[i * (nbatch+windowBlocks-1) * nchan * 2], 
	&cuda_inp_buf[i*(nbatch+windowBlocks-1)*nchan*2 + (nbatch)*nchan*2], 
	(windowBlocks-1) * nchan * 2 * sizeof(float), hipMemcpyDeviceToDevice );
  
  /* Thread number should be multiple of 32 for best efficiency */
  /* Assume nchan to be power of 2 and larger than numThreads */
  dim3 threads( numThreads, 1, 1 );
  dim3 blocks( nchan * 2 / numThreads, nbatch, ninp );
  
  /* cuda_inp_buf needs to be offset by (windowBlocks-1) chunks due to the algorithm design */
  gettimeofday( &thetime, NULL );
  if( wordtype == 0 )
    unpackUnsignedData_kernel<<< blocks, threads >>>(cudaBuffer, &cuda_inp_buf[(windowBlocks-1) * nchan * 2]);
  /* FIXME: Not sure about the correctness of signed data unpacking */
  else if( wordtype == 1 )
    unpackSignedData_kernel<<< blocks, threads >>>(cudaBuffer, &cuda_inp_buf[(windowBlocks-1) * nchan * 2]);

  hipDeviceSynchronize();
  unpackTime += elapsed_time(&thetime);
  totalTime += elapsed_time(&starttime);

  //fprintf( stderr, "hipMemcpy time: %g, size: %d MB\n", cudaCopyTime, ntoread / 1024 / 1024 );

  return 0;
}


/* Calculate the polyphase output using the method of choice */
void polyphase_gpu(int ninp, int windowBlocks, int size, int nbatch, 
    char *polyMethod, float *cuda_poly_buf, float *cuda_inp_buf, float *cuda_window_buf)
{
  int numThreads = 128;
  hipMemset( cuda_poly_buf, 0, ninp * size * nbatch * sizeof(float) );

  /* Polyphase calculation by adding up the weighted time segments */
  if( strcmp(polyMethod, "weighted-overlap-add") == 0 )
  {
    /* Thread numbers should be multiple of 32 for best efficiency */
    /* Assume also windowBlocks is a power of 2 */
    dim3 threads( numThreads, 1, 1 );
    dim3 blocks( size / numThreads, nbatch, ninp );
    overlap_add_kernel<<< blocks, threads >>>(cuda_poly_buf, cuda_inp_buf, 
	cuda_window_buf, windowBlocks);
  }

  /* Polyphase calculation by performing FFT at higher sample rate and decimate */
  /* FIXME: Not yet implemented */
  else if( strcmp(polyMethod, "oversample-decimate") == 0 )
  {
    dim3 threads( numThreads, 1, 1 );
    dim3 blocks( size / numThreads, nbatch, ninp );
    oversample_decimate_kernel<<< blocks, threads >>>(cuda_poly_buf, cuda_inp_buf, 
	cuda_window_buf, windowBlocks);
  }
  else
  {
    fprintf( stderr, "Invalid polyphase method: %s\n", polyMethod );
    exit(1);
  }
}

/* CUDA FFT, will perform parallel execution if ninp > 1 */
void do_CUFFT(int nchan, int ninp, int nbatch, float *cuda_poly_buf, hipfftComplex *cuda_ft_buf)
{
  static hipfftHandle plan;
  static int doneplan = 0;

  if( !doneplan ) 
  {
    /* Setup the FFT plan for CUDA, it will do parallel FFT if ninp > 1 */
    hipfftPlan1d( &plan, nchan * 2, HIPFFT_R2C, ninp * nbatch );
    doneplan = 1;
  }

  hipfftExecR2C( plan, cuda_poly_buf, cuda_ft_buf );
}

/* Perform CMAC in GPU. */
void do_CUDA_CMAC(int nchan, int ninp, int ncross, int nbatch, int prod_type, hipfftComplex *cuda_ft_buf, hipfftComplex *cuda_cross_corr, float *cuda_auto_corr)
{
  int numThreads = 64;

  if( prod_type == 'A' || prod_type == 'B' )
  {
    /* Uses ninp instead of ncorr for auto correlator */
    dim3 threads( numThreads, 1, 1 );
    dim3 blocks( nchan / numThreads, ninp, 1 );
    CMAC_auto_kernel<<< blocks, threads >>>(nbatch, cuda_ft_buf, cuda_auto_corr);
  }
  else if( prod_type == 'C' || prod_type == 'B' )
  {
    /* Cannot do cross correlation with only 1 input stream */
    if( ninp == 1 || ncross == 0 ) 
    {
      fprintf( stderr, "Warning: attempt to perform cross correlation with only 1 input stream.\n" );
    }
    else
    {
      /* Cross correlator using ncorr */
      dim3 threads( numThreads, 1, 1);
      dim3 blocks( nchan / numThreads, ncross, 1 );

      CMAC_cross_kernel<<< blocks, threads >>>(ninp, nbatch, cuda_ft_buf, cuda_cross_corr);
    }
  }
  else
  {
    fprintf( stderr, "Invalid prod_type\n" );
    exit(1);
  }
}

/* Write the results into a buffer of output, size set to
 yaxis_size, the whole output will be written to a file everytime 
 it obtains nrows_per_refresh output. yaxis_size must be divisible
 by rows_per_refresh */
void writeGPUOutput(FILE *fout_ac, FILE *fout_cc, int ninp, int nchan, 
    int ncross, int naver, int prod_type, int nbatch, int isLast, 
    float normaliser, int yaxis_size, int rows_per_refresh,
    hipfftComplex *cuda_cross_corr, float *cuda_auto_corr)
{
  int i;
  static int init = 1;
  static complex float *ctemp_buf;
  static float *temp_buf = NULL;

  static int row = 0;
  static int rowBlock= 0;
  static int nrowBlocks = yaxis_size / rows_per_refresh;
  static FILE *ftemp_ac;
  static FILE *ftemp_cc;

  char filename[BUFSIZ];

  //static FILE *fp;

  int numThreads = 32;

  if( init )
  {
    //fp = fopen( "temp.csv", "w" );

    init = 0;
    ctemp_buf = (complex float *)malloc( yaxis_size * nchan * ncross * sizeof(float) );
    temp_buf = (float *)malloc( yaxis_size * nchan * ninp * sizeof(float) );
  }

  dim3 threads(numThreads, 1, 1);
  dim3 blocks(nchan / numThreads, ninp, 1);

  /* Keep track of which row is it now */
  /* rowBlock is used to keep track of which block of data is getting printed out, when row = 10-19, rowBlock = 1 (if rows_per_refresh = 10) */
  rowBlock = row / rows_per_refresh; 
  
  if( prod_type == 'A' || prod_type == 'B' )
  {
    /* Normalisation, number of points are obtained from the threads and blocks number */
    normalise_float_kernel<<< blocks, threads >>>(cuda_auto_corr, normaliser);
    
    /* Extract the real numbers, for auto correlation only */
    hipMemcpy( &temp_buf[row*nchan*ninp], cuda_auto_corr, nchan * ninp * sizeof(float), hipMemcpyDeviceToHost );
    
    /*for( i = 0; i < nchan * ninp; i++ )
    {
      fprintf( fp, "%e ", 10 * log(temp_buf[i]) );
    }
    fprintf( fp, "\n" );*/
    fwrite( &temp_buf[row*nchan*ninp], sizeof(float), nchan * ninp, fout_ac );
    hipMemset( cuda_auto_corr, 0, (nchan) * ninp * sizeof(float) );
  
    /* output auto correlation results to a file with time stamp */
    if( row % rows_per_refresh == 0 )
    {
      time_stamp(filename);
      /* ac extension to mean auto correlation */
      sprintf( filename, "%s.ac", filename );

      ftemp_ac = fopen( filename, "w" );

      for( i = (rowBlock+1)%nrowBlocks; i != rowBlock; i = (i+1)%nrowBlocks )
      {
	fwrite( &temp_buf[i*rows_per_refresh*nchan*ninp], sizeof(float), 
	    rows_per_refresh*nchan*ninp, ftemp_ac );
      }

      fclose(ftemp_ac);
    }

  }

  else if( prod_type == 'C' || prod_type == 'B' )
  {
    /* Just change the y-dimension of the block number */
    blocks.y = ncross;
    normalise_complex_kernel<<< blocks, threads >>>(cuda_cross_corr, normaliser);
    
    hipMemcpy( &ctemp_buf[row*nchan*ncross], cuda_cross_corr, (nchan) * ncross * sizeof(complex float), hipMemcpyDeviceToHost );

    fwrite( &ctemp_buf[row*nchan*ncross], sizeof(complex float), nchan * ncross, fout_cc );
    hipMemset( cuda_cross_corr, 0, (nchan) * ncross * sizeof(hipfftComplex) );

    /* output cross correlation results to a file with time stamp */
    if( row % rows_per_refresh == 0 )
    {
      time_stamp(filename);
      /* ac extension to mean auto correlation */
      sprintf( filename, "%s.cc", filename );

      ftemp_cc = fopen( filename, "w" );

      for( i = (rowBlock+1)%nrowBlocks; i != rowBlock; i = (i+1)%nrowBlocks )
      {
	fwrite( &ctemp_buf[i*rows_per_refresh*nchan*ninp], sizeof(hipfftComplex), 
	    rows_per_refresh*nchan*ninp, ftemp_cc );
      }


      fclose(ftemp_cc);
    }
  }

  else
  {
    fprintf( stderr, "Invalid prod type: %c\n", (char) prod_type );
    exit(1);
  }

  row = (row+1) % yaxis_size;

  /* When this is the last output, free the memory */
  if( isLast )
  {
    free(temp_buf);
    free(ctemp_buf);
  }
}

/* returns the elapsed wall-clock time, in ms, since start (without resetting start) */
float elapsed_time(struct timeval *start){
    struct timeval now;
    gettimeofday(&now,NULL);
    return 1.e3f*(float)(now.tv_sec-start->tv_sec) +
        1.e-3f*(float)(now.tv_usec-start->tv_usec);
}

/* return a string representation of local time stamp, in the format of
 yyyy-mm-dd-hh:mm:ss */
void time_stamp(char *str)
{
  struct tm *tm;
  time_t current;

  time(&current);

  tm = localtime(&current);

  /* Assume that the char array is large enough */
  sprintf( str, "%d-%.2d-%.2d-%.2d:%.2d:%.2d", 
      1900 + tm->tm_year, tm->tm_mon, tm->tm_mday, 
      tm->tm_hour, tm->tm_min, tm->tm_sec, str );
}
